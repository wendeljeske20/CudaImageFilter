#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "CImg.h"
#include <math.h>  

#include <iostream>
#include <string>
using namespace std;
using namespace cimg_library;


void ProcessImage(unsigned int *defaultColor, unsigned int *newColor, int imageWidth, int imageHeight, float *filter, int filterWidth, float filterFactor);

__host__ __device__ void PackColorBits(unsigned int &color, int r, int g, int b)
{
	color = ((r & 0x0ff) << 16) | ((g & 0x0ff) << 8) | (b & 0x0ff);
}

__host__ __device__ void UnpackColorBits(int rgb, int &r, int &g, int &b)
{
	r = (rgb >> 16) & 0x0ff;
	g = (rgb >> 8) & 0x0ff;
	b = (rgb) & 0x0ff;
}

void HostConvolutionFilter(unsigned int *defaultColor, unsigned int *newColor, int imageWidth, int imageHeight, float *filter, int filterWidth, float filterFactor)
{
	int r, g, b;

	const int midX = floor(float(filterWidth));
	const int midY = floor(float(filterWidth));

	//float factor = 1.0;
	float bias = 0;



	//apply the filter
	int imageSize = imageWidth * imageHeight;
	for (int i = 0; i < imageSize; i++)
	{


		float red = 0.0, green = 0.0, blue = 0.0;


		for (int k = 0; k < filterWidth * filterWidth; k++)
		{

			//printf("for");
			int filterX;
			if (k > 0)
				filterX = k % filterWidth;
			else
				filterX = 0;

			int filterY;
			if (k > 0)
				filterY = floor(float(k) / filterWidth);
			else
				filterY = 0;


			int pixelOffset = filterX - midX + imageWidth * (filterY - midY);
			int index = i + pixelOffset;

			//int index = min(max(i + pixelOffset, 0), 1360 * 7 - 1);
			//int index = 1360 * ();
			if (index >= 0 && index < imageSize)
			{
				UnpackColorBits(defaultColor[index], r, g, b);
				//cout << r << " ";
				red += r * filter[k];
				green += g * filter[k];
				blue += b * filter[k];
			}
		}

		//truncate values smaller than zero and larger than 255
		r = min(max(int(filterFactor * red + bias), 0), 255);
		g = min(max(int(filterFactor * green + bias), 0), 255);
		b = min(max(int(filterFactor * blue + bias), 0), 255);

		//defaultColor[i] = 200;
		PackColorBits(newColor[i], r, g, b);
	}
}

__global__ void DeviceConvolutionFilter(unsigned int *defaultColor, unsigned int *newColor, int *imageWidth, int *imageHeight, float *filter, int *filterWidth, float *filterFactor)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int r, g, b;

	const int midX = floor(float(*filterWidth));
	const int midY = floor(float(*filterWidth));

	float bias = 0;

	float red = 0.0, green = 0.0, blue = 0.0;

	int filterSize = *filterWidth * *filterWidth;

	for (int k = 0; k < filterSize; k++)
	{

		//printf("for");
		int filterX;
		if (k > 0)
			filterX = k % *filterWidth;
		else
			filterX = 0;

		int filterY;
		if (k > 0)
			filterY = floor(float(k) / *filterWidth);
		else
			filterY = 0;


		int pixelOffset = filterX - midX + *imageWidth * (filterY - midY);
		int index = i + pixelOffset;
		//int index = 1360 * ();

		UnpackColorBits(defaultColor[index], r, g, b);
		red += r * filter[k];
		green += g * filter[k];
		blue += b * filter[k];

	}

	//truncate values smaller than zero and larger than 255
	r = min(max(int(*filterFactor * red + bias), 0), 255);
	g = min(max(int(*filterFactor * green + bias), 0), 255);
	b = min(max(int(*filterFactor * blue + bias), 0), 255);


	PackColorBits(newColor[i], r, g, b);
}







void SetImageColor(CImg<unsigned char> &image, unsigned int *color, int width, int height)
{
	int k = 0;

	for (int y = 0; y < height; y++)
	{

		for (int x = 0; x < width; x++)
		{
			//cor[y * width + x] = 225125050;
			//newImage[y * width + x] = 2000;

			int r, g, b;
			UnpackColorBits(color[k], r, g, b);

			image(x, y, 0) = r;
			image(x, y, 1) = g;
			image(x, y, 2) = b;
			//if(k < 5000)
			k++;
		}
	}
}

void SetColorArray(unsigned int *color, CImg<unsigned char> image, int width, int height)
{
	for (int x = 0; x < width; x++)
	{
		for (int y = 0; y < height; y++)
		{
			PackColorBits(color[y * width + x], image(x, y, 0), image(x, y, 1), image(x, y, 2));
		}
	}
}

int main()
{

	int imageWidth = 1360;
	int imageHeight = 768;

	CImg<unsigned char> image1("printdoinvicto.bmp");
	CImg<unsigned char> image2("astroneer.bmp");
	CImg<unsigned char> newImage(imageWidth, imageHeight, 1, 3, 255);// = image;

	

	float motionBlurFilter[] =
	{
	  1, 0, 0, 0, 0, 0, 0, 0, 0,
	  0, 1, 0, 0, 0, 0, 0, 0, 0,
	  0, 0, 1, 0, 0, 0, 0, 0, 0,
	  0, 0, 0, 1, 0, 0, 0, 0, 0,
	  0, 0, 0, 0, 1, 0, 0, 0, 0,
	  0, 0, 0, 0, 0, 1, 0, 0, 0,
	  0, 0, 0, 0, 0, 0, 1, 0, 0,
	  0, 0, 0, 0, 0, 0, 0, 1, 0,
	  0, 0, 0, 0, 0, 0, 0, 0, 1,
	};

	float edgeEnhancementFilter[] =
	{
		 0,-1, 0,
		-1, 1, 0,
		 0, 0, 0
	};

	float sharpenFilter[] =
	{
		 0,-1, 0,
		-1, 5,-1,
		 0,-1, 0
	};

	float embossFilter[] =
	{
		-2,-1, 0,
		-1, 1, 1,
		 0, 1, 2
	};

	const int filterWidth = 3;
	float *currentFilter = sharpenFilter;

	float filterFactor = 1.0;


	//as cores s�o armazenas em um array de inteiros unidimensional usando empacotamento de bits
	unsigned int *color = new unsigned int[imageWidth * imageHeight];
	unsigned int *newColor = new unsigned int[imageWidth * imageHeight];

	for (int i = 0; i < imageWidth * imageHeight; i++)
	{
		newColor[i] = 100;
	}

	//SetColorArray(color, image1, width, height);
	//ProcessImage(color, width, height, filter, filterWidth, filterFactor);
	//SetImageColor(newImage, color, width, height);

	CImgDisplay window(imageWidth, imageHeight);

	CImg<unsigned char> imgtext;
	unsigned char clr[] = { 255,0,0 };




	while (!window.is_closed())
	{


		if (window.is_keyESC())
			window.close();


		if (window.is_keyARROWRIGHT())
		{
			filterFactor *= 1.1f;
			//printf("%f ",filterFactor);
		}
		else if (window.is_keyARROWLEFT())
		{
			filterFactor *= 0.9f;
			//printf("%f ", filterFactor);
		}


		if (window.is_keyC())
		{
			SetColorArray(color, image1, imageWidth, imageHeight);
			HostConvolutionFilter(color, newColor, imageWidth, imageHeight, currentFilter, filterWidth, filterFactor);
			SetImageColor(newImage, newColor, imageWidth, imageHeight);
		}

		if (window.is_keyG())
		{
			SetColorArray(color, image2, imageWidth, imageHeight);
			ProcessImage(color, newColor, imageWidth, imageHeight, currentFilter, filterWidth, filterFactor);
			SetImageColor(newImage, newColor, imageWidth, imageHeight);

		}






		window.display(newImage);

		/*string txt = to_string(filterFactor);
		imgtext.draw_text(2000, 2000, txt.c_str(), clr, 0, 1, 23);
		window.display(imgtext);*/

		//window.
		//imgtext.clear();
		window.wait();
	}


	//newImage.save("printdoinvictocomfiltro.bmp");


	return 0;
}




void ProcessImage(unsigned int *defaultColor, unsigned int *newColor, int imageWidth, int imageHeight, float *filter, int filterWidth, float filterFactor)
{

	unsigned int *deviceColor = 0;
	unsigned int *deviceNewColor = 0;
	int *deviceImageWidth = 0;
	int *deviceImageHeight = 0;
	float *deviceFilter = 0;
	int *deviceFilterWidth = 0;
	float *deviceFilterFactor = 0;


	unsigned int imageSize = imageWidth * imageHeight;


	// Allocate GPU buffers 
	hipMalloc((void**)&deviceColor, imageSize * sizeof(int));
	hipMalloc((void**)&deviceNewColor, imageSize * sizeof(int));
	hipMalloc((void**)&deviceImageWidth, sizeof(int));
	hipMalloc((void**)&deviceImageHeight, sizeof(int));

	hipMalloc((void**)&deviceFilter, filterWidth * filterWidth * sizeof(float));
	hipMalloc((void**)&deviceFilterWidth, sizeof(int));
	hipMalloc((void**)&deviceFilterFactor, sizeof(int));


	// Copy from host memory to GPU buffers.
	hipMemcpy(deviceColor, defaultColor, imageSize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceImageWidth, &imageWidth, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceImageHeight, &imageHeight, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(deviceFilter, filter, filterWidth * filterWidth * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceFilterWidth, &filterWidth, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceFilterFactor, &filterFactor, sizeof(int), hipMemcpyHostToDevice);


	// Launch a kernel on the GPU with one thread for each element.
	DeviceConvolutionFilter << <imageSize / 512, 512 >> > (deviceColor, deviceNewColor, deviceImageWidth, deviceImageWidth, deviceFilter, deviceFilterWidth, deviceFilterFactor);


	hipDeviceSynchronize();

	// Copy from GPU buffer to host memory.
	hipMemcpy(newColor, deviceNewColor, imageSize * sizeof(int), hipMemcpyDeviceToHost);


	hipFree(deviceColor);
	hipFree(deviceNewColor);
	hipFree(deviceImageWidth);
	hipFree(deviceImageHeight);
	hipFree(deviceFilter);
	hipFree(deviceFilterWidth);
	hipFree(deviceFilterFactor);

}


